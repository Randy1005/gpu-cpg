#include "hip/hip_runtime.h"
#include "gpucpg.cuh"
#include <cassert>

int main(int argc, char* argv[]) {
  if (argc != 6) {
    std::cerr << "usage: ./a.out [benchmark] [k] [enable_csr_reorder_gpu] [enable_fused_steps] [enable_runtime_measure]\n";
    std::exit(1);
  }

  std::string benchmark = argv[1];
  auto num_paths = std::stoi(argv[2]);
  bool enable_csr_reorder_gpu = std::stoi(argv[3]);
  bool enable_fused_steps = std::stoi(argv[4]);
  bool enable_interm_perf_log = std::stoi(argv[5]);
  int max_dev_lvls{5};
  bool enable_compress{true};

  gpucpg::CpGen 
    cpgen_lvlize_td_then_relax_bu, 
    cpgen_lvlize_td_then_relax_bu_reindex,
    cpgen_ref;

  #pragma omp parallel
  #pragma omp single
  {
    #pragma omp task
    cpgen_lvlize_td_then_relax_bu.read_input(benchmark);
    
    #pragma omp task
    cpgen_lvlize_td_then_relax_bu_reindex.read_input(benchmark); 
    
    #pragma omp task
    cpgen_ref.read_input(benchmark);
  }
  #pragma omp taskwait

  
  std::ofstream runtime_log_file(benchmark+"-rt.log");
  int N = cpgen_lvlize_td_then_relax_bu.num_verts();
  int M = cpgen_lvlize_td_then_relax_bu.num_edges();
  runtime_log_file << "==== Runtime Log for benchmark: " 
                   << benchmark 
                   << " (N=" << N 
                   << ", M=" << M
                   << ", num_paths=" << num_paths
                   << ") ====\n";
  
  // cpgen_ref.report_paths(num_paths, max_dev_lvls, enable_compress,
  //   gpucpg::PropDistMethod::LEVELIZE_THEN_RELAX, gpucpg::PfxtExpMethod::SEQUENTIAL);
  
  // std::cout << "==================================\n";
  // auto slks_ref = cpgen_ref.get_slacks(num_paths);
  // std::cout << "REF: " << "last slack=" << slks_ref.back() << "\n";
  // std::cout << "pfxt expansion time=" << cpgen_ref.expand_time / 1ms << " ms.\n";
  std::chrono::duration<double, std::micro> total_lvlize_time{0};
  std::chrono::duration<double, std::micro> total_prefix_scan_time{0};
  std::chrono::duration<double, std::micro> total_csr_reorder_time{0};
  std::chrono::duration<double, std::micro> total_relax_time{0};
  std::chrono::duration<double, std::micro> total_pfxt_time{0};
  for (int run = 0; run < 10; run++) {
    cpgen_lvlize_td_then_relax_bu.report_paths(num_paths, max_dev_lvls, enable_compress,
      gpucpg::PropDistMethod::LEVELIZE_THEN_RELAX, gpucpg::PfxtExpMethod::SHORT_LONG,
      false, 0.005f, 5.0f, 8, false, false, false, enable_interm_perf_log);
    total_lvlize_time += cpgen_lvlize_td_then_relax_bu.lvlize_time;
    total_relax_time += cpgen_lvlize_td_then_relax_bu.relax_time;
    total_pfxt_time += cpgen_lvlize_td_then_relax_bu.expand_time; 
    cpgen_lvlize_td_then_relax_bu.reset();
  }
  runtime_log_file 
    << "==== No CSR reorder ====\n"
    << "Total Levelize Time (avg): " << total_lvlize_time/1ms/10.0f << " ms.\n"
    << "Total Relax Time (avg): " << total_relax_time/1ms/10.0f << " ms.\n"
    << "Total Pfxt Expansion Time (avg): " << total_pfxt_time/1ms/10.0f << " ms.\n";

  // std::cout << "==================================\n";  
  // auto slks_lvlize_td_then_relax = cpgen_lvlize_td_then_relax_bu.get_slacks(num_paths);
  // std::cout << "LEVELIZE_THEN_RELAX: " <<  "last slack=" << slks_lvlize_td_then_relax.back() << "\n";
  // std::cout << "LEVELIZE_THEN_RELAX runtime=" << cpgen_lvlize_td_then_relax_bu.prop_time / 1ms << " ms.\n";
  // std::cout << "pfxt expansion time=" << cpgen_lvlize_td_then_relax_bu.expand_time / 1ms << " ms.\n";

  // reset the timings
  total_lvlize_time = std::chrono::duration<double, std::micro>{0};
  total_prefix_scan_time = std::chrono::duration<double, std::micro>{0};
  total_csr_reorder_time = std::chrono::duration<double, std::micro>{0};
  total_relax_time = std::chrono::duration<double, std::micro>{0};
  total_pfxt_time = std::chrono::duration<double, std::micro>{0};

  for (int run = 0; run < 10; run++) {
    cpgen_lvlize_td_then_relax_bu_reindex.report_paths(num_paths, max_dev_lvls, enable_compress,
      gpucpg::PropDistMethod::LEVELIZE_THEN_RELAX, gpucpg::PfxtExpMethod::SHORT_LONG, 
      false, 0.005f, 5.0f, 8, false, enable_csr_reorder_gpu, enable_fused_steps, enable_interm_perf_log);
      
    total_lvlize_time += cpgen_lvlize_td_then_relax_bu_reindex.lvlize_time;
    total_prefix_scan_time += cpgen_lvlize_td_then_relax_bu_reindex.prefix_scan_time;
    total_csr_reorder_time += cpgen_lvlize_td_then_relax_bu_reindex.csr_reorder_time;
    total_relax_time += cpgen_lvlize_td_then_relax_bu_reindex.relax_time;
    total_pfxt_time += cpgen_lvlize_td_then_relax_bu_reindex.expand_time; 

    cpgen_lvlize_td_then_relax_bu_reindex.reset();
  }

  runtime_log_file 
    << "==== With CSR reorder (GPU) ====\n"
    << "Total Levelize Time (avg): " << total_lvlize_time/1ms/10.0f << " ms.\n"
    << "Total Prefix Scan Time (avg): " << total_prefix_scan_time/1ms/10.0f << " ms.\n"
    << "Total CSR Reorder Time (avg): " << total_csr_reorder_time/1ms/10.0f << " ms.\n"
    << "Total Relax Time (avg): " << total_relax_time/1ms/10.0f << " ms.\n"
    << "Total Pfxt Expansion Time (avg): " << total_pfxt_time/1ms/10.0f << " ms.\n";
  

  // std::cout << ========================\n";
  // auto slks_lvlize_td_then_relax_reindex = cpgen_lvlize_td_then_relax_bu_reindex.get_slacks(num_paths);
  // std::cout << "LEVELIZE_THEN_RELAX_REINDEX: " << "last slack=" << slks_lvlize_td_then_relax_reindex.back() << "\n";
  // std::cout << "LEVELIZE_THEN_RELAX_REINDEX runtime=" << cpgen_lvlize_td_then_relax_bu_reindex.prop_time / 1ms << " ms.\n";
  // std::cout << "pfxt expansion time=" << cpgen_lvlize_td_then_relax_bu_reindex.expand_time / 1ms << " ms.\n";
  
  return 0;
}